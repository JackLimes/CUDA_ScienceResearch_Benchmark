
#include "hip/hip_runtime.h"

#include <ctime>
#include <stdio.h>
#include <stdlib.h>






__global__ void agg(int *b, unsigned int size) {
	clock_t begin = clock();
	while (begin - clock() < 1000) {
		b[blockIdx.x] = b[blockIdx.x] + 1;
	}
}

int main(void) {
	printf("Running...");

	//Hailiang Zhang's device counting code
	int deviceCount, device;
	int gpuDeviceCount = 0;
	short threads = 0;
	struct hipDeviceProp_t properties;
	hipError_t cudaResultCode = hipGetDeviceCount(&deviceCount);
	if (cudaResultCode != hipSuccess)
		deviceCount = 0;
	/* machines with no GPUs can still report one emulation device */
	for (device = 0; device < deviceCount; ++device) {
		hipGetDeviceProperties(&properties, device);
		if (properties.major != 9999) /* 9999 means emulation only */
			if (device == 0)
			{
				printf("multiProcessorCount %d\n", properties.multiProcessorCount);
				printf("maxThreadsPerMultiProcessor %d\n", properties.maxThreadsPerMultiProcessor);
				
			}
	}
	threads = properties.multiProcessorCount;
	printf("%d", threads);
	//talk about how N must for some reason be a constant value. Must be manually changed for each graphics card.
	const short N = 15;
	int b[N] = { 0 };
	int *d_b;
	hipSetDevice(0);

	//while (clock() - begin < 1000){
		// Allocate space for device copies of  b
		hipMalloc((void **)&d_b, N * sizeof(int));

		// Copy inputs to device
		hipMemcpy(d_b, &b, N * sizeof(int), hipMemcpyHostToDevice);

		agg <<<N, 1 >>> (d_b, N);

		// Copy result back to host
		hipMemcpy(&b, d_b, N * sizeof(int), hipMemcpyDeviceToHost);

		// Cleanup
		hipFree(d_b);

		
		
		//getchar();
	//}


	printf("Result: %d per element, with %d threads in pararallel leads to a final score of...\n%d", b[0], N, (b[0]* N));
	getchar();
	hipDeviceReset();
	return 0;

}
