#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <ctime>
#include <stdio.h>
#include <stdlib.h>
#include<iostream>
using namespace std;




__global__ void agg(int *b, unsigned int size) {
		b[blockIdx.x] = b[blockIdx.x] + 1;
}

int main(void) {

	//Hailiang Zhang's device counting code
	int deviceCount, device;
	short threads = 0;
	struct hipDeviceProp_t properties;
	hipError_t cudaResultCode = hipGetDeviceCount(&deviceCount);
	if (cudaResultCode != hipSuccess)
		deviceCount = 0;
	/* machines with no GPUs can still report one emulation device */
	for (device = 0; device < deviceCount; ++device) {
		hipGetDeviceProperties(&properties, device);
		if (properties.major != 9999) /* 9999 means emulation only */
			if (device == 0)
			{
				printf("multiProcessorCount %d\n", properties.multiProcessorCount);
				printf("maxThreadsPerMultiProcessor %d\n", properties.maxThreadsPerMultiProcessor);
				
			}
	}
	threads = properties.multiProcessorCount;
	printf("%d parallel blocks (because your gpu can support it!)\n", threads);

	//talk about how N must for some reason be a constant value. Must be manually changed for each graphics card.
	const short N = threads;
	int b[100] = { 0 }; //make this array so big there's no way any gpu will go over it.
	int *d_b;
	int span;
	_int64 result;
	clock_t begin = clock();

	hipSetDevice(0);

	// Allocate space for device copies of  b and begin
	hipMalloc((void **)&d_b, N * sizeof(int));

	// Copy inputs to device
	hipMemcpy(d_b, &b, N * sizeof(int), hipMemcpyHostToDevice);
	printf("How long do you wish to run this test in milliseconds? (1 second = 1000 Milliseconds) "); cin >> span;
	printf("Running...\n");
	while(clock() - begin < span) {
		agg << <N, 1 >> > (d_b, N);
	}

	// Copy result back to host
	hipMemcpy(&b, d_b, N * sizeof(int), hipMemcpyDeviceToHost);

	// Cleanup
	hipFree(d_b);
		

	result = b[0] * N;
	printf("Result: %d per element, with %d threads in pararallel leads to a final score of...\n%d", b[0], N, result);
	printf("\n");
	printf("{");
	for (int i = 0; i < N -1; i++) {
		printf("%d, ", b[i]);
	}
	printf("%d}\n", b[N - 1]);
	hipDeviceReset();
	system("pause");
	return 0;

}
